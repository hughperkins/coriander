
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    int num=25;
    hipDeviceptr_t loc[num];
    for(int i=0; i<num; i++){
	std::cout << "i: " << i << std::endl;
        hipMalloc(&loc[i], i);
    }
    for(int i=0; i<num; i++){
        hipFree(loc[i]);
    }
    return 0;
}
