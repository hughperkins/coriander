// double indirection, ie float **, in kernel parameter

// this test cuts all gpu buffers from one single gpu buffer

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct BoundedArray {
    float *bounded_array[8];
};

__global__ void wipe(int *buffer, int length) {
    length >>= 5;
    int tid = threadIdx.x;
    for(int i = 0; i < length; i++) {
        buffer[(i << 5) + tid] = -1;
    }
}

__global__ void run_bounded_array(struct BoundedArray boundedArray, int numBuffers, int N) {
    for(int i = 0; i < numBuffers; i++) {
        for(int j = 0; j < N; j++) {
            boundedArray.bounded_array[i][j] = 123.0f + i + 1 + j;
        }
    }
}

void test1() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    const int numBuffers = 3;

    char *gpuArena;
    int mallocSize = numBuffers * N * sizeof(float) + 256 + 1024;
    std::cout << "mallocSize=" << mallocSize << std::endl;
    hipMalloc((void **)&gpuArena, mallocSize);
    wipe<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((int *)gpuArena, mallocSize >> 2);

    struct BoundedArray boundedArray;
    float *hostFloats[numBuffers];

    for(int i = 0; i < numBuffers; i++) {
        boundedArray.bounded_array[i] = (float *)(gpuArena + 256 + i * N * sizeof(float));
        std::cout << "bounded_array[" << i << "]=" << (long)boundedArray.bounded_array[i] << std::endl;
        hostFloats[i] = new float[N];
    }

    run_bounded_array<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(boundedArray, numBuffers, N);

    for(int i = 0; i < numBuffers; i++) {
        hipMemcpy(hostFloats[i], boundedArray.bounded_array[i], N * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipStreamSynchronize(stream);

    std::cout << std::endl;
    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < 4; j++) {
            cout << hostFloats[i][j] << " ";
        }
        cout << endl;
    }

    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < N; j++) {
            float expected = 123.0f + 1 + i + j;
            float actual = hostFloats[i][j];
            if(actual != expected) {
                std::cout << "mismatch for i=" << i << " j=" << j << " expected=" << expected << " actual=" << actual << std::endl;
                assert(false);
            }
        }
    }

    for(int i=0; i < numBuffers; i++) {
        delete[] hostFloats[i];
    }
    hipFree(gpuArena);

    hipStreamDestroy(stream);
    std::cout << "test1 finished ok" << std::endl;
}

struct BoundedArrayUnion {
    float *bounded_array[8];
    float **unbounded_array;
};

__global__ void run_bounded_array_two(struct BoundedArrayUnion mystruct, int useUnbounded, int numBuffers, int N) {
    float **starstar = useUnbounded ? mystruct.unbounded_array : mystruct.bounded_array;
    for(int i = 0; i < numBuffers; i++) {
        for(int j = 0; j < N; j++) {
            starstar[i][j] = 123.0f + i + 1 + j;
        }
    }
}

void test2_bounded() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    const int numBuffers = 3;

    char *gpuArena;
    int mallocSize = numBuffers * N * sizeof(float) + 256 + 1024;
    std::cout << "mallocSize=" << mallocSize << std::endl;
    hipMalloc((void **)&gpuArena, mallocSize);
    wipe<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((int *)gpuArena, mallocSize >> 2);

    struct BoundedArrayUnion boundedArray;
    float *hostFloats[numBuffers];

    for(int i = 0; i < numBuffers; i++) {
        boundedArray.bounded_array[i] = (float *)(gpuArena + 256 + i * N * sizeof(float));
        std::cout << "bounded_array[" << i << "]=" << (long)boundedArray.bounded_array[i] << std::endl;
        hostFloats[i] = new float[N];
    }

    run_bounded_array_two<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(boundedArray, 0, numBuffers, N);

    for(int i = 0; i < numBuffers; i++) {
        hipMemcpy(hostFloats[i], boundedArray.bounded_array[i], N * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipStreamSynchronize(stream);

    std::cout << std::endl;
    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < 4; j++) {
            cout << hostFloats[i][j] << " ";
        }
        cout << endl;
    }

    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < N; j++) {
            float expected = 123.0f + 1 + i + j;
            float actual = hostFloats[i][j];
            if(actual != expected) {
                std::cout << "mismatch for i=" << i << " j=" << j << " expected=" << expected << " actual=" << actual << std::endl;
                assert(false);
            }
        }
    }

    for(int i=0; i < numBuffers; i++) {
        delete[] hostFloats[i];
    }
    hipFree(gpuArena);

    hipStreamDestroy(stream);
    std::cout << "finished test2_bounded" << std::endl;
}

void test3_unbounded() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    const int numBuffers = 3;

    char *gpuArena;
    int mallocSize = numBuffers * N * sizeof(float) + 256 + 1024;
    std::cout << "mallocSize=" << mallocSize << std::endl;
    hipMalloc((void **)&gpuArena, mallocSize);
    wipe<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((int *)gpuArena, mallocSize >> 2);

    struct BoundedArrayUnion boundedArray;
    float *hostFloats[numBuffers];

    for(int i = 0; i < numBuffers; i++) {
        boundedArray.bounded_array[i] = (float *)(gpuArena + 256 + i * N * sizeof(float));
        std::cout << "bounded_array[" << i << "]=" << (long)boundedArray.bounded_array[i] << std::endl;
        hostFloats[i] = new float[N];
    }

    run_bounded_array_two<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(boundedArray, 1, numBuffers, N);

    for(int i = 0; i < numBuffers; i++) {
        hipMemcpy(hostFloats[i], boundedArray.bounded_array[i], N * sizeof(float), hipMemcpyDeviceToHost);
    }
    hipStreamSynchronize(stream);

    std::cout << std::endl;
    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < 4; j++) {
            cout << hostFloats[i][j] << " ";
        }
        cout << endl;
    }

    for(int i = 0; i < numBuffers; i++) {
        for(int j=0; j < N; j++) {
            float expected = 123.0f + 1 + i + j;
            float actual = hostFloats[i][j];
            if(actual != expected) {
                std::cout << "mismatch for i=" << i << " j=" << j << " expected=" << expected << " actual=" << actual << std::endl;
                assert(false);
            }
        }
    }

    for(int i=0; i < numBuffers; i++) {
        delete[] hostFloats[i];
    }
    hipFree(gpuArena);

    hipStreamDestroy(stream);
    std::cout << "finished test3_unbounded" << std::endl;
}

int main(int argc, char *argv[]) {
    // test1();
    // test2_bounded();
    test3_unbounded();
    return 0;
}
