/*
other things we should test:
- struct pointer, with offset
- multiple struct pointers, cut from same buffer
- getting values from various types of structs passed in
*/

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct Struct_fp_fp_f_f {
    float *p1;
    float *p2;
    float f1;
    float f2;
};

struct Struct_fp {
    float *p1;
};

struct Struct_1float {
    float f1;
};

struct Struct_2floats {
    float f1;
    float f2;
};

// __global__ void struct_byvalue(struct Struct_fp_fp_f_f mystruct, float *out) {
//     out[0] = mystruct.f1;
//     out[1] = mystruct.f2;
//     mystruct.p1[0] = 9.0f;
//     mystruct.p2[0] = 10.0f;
// }

// void testbyvaluestruct() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     float *gpuFloats2;
//     cudaMalloc((void**)(&gpuFloats2), N * sizeof(float));

//     float *gpuFloats3;
//     cudaMalloc((void**)(&gpuFloats3), N * sizeof(float));

//     float *gpuOut;
//     cudaMalloc((void**)(&gpuOut), N * sizeof(float));

//     float *hostFloats1 = new float[N];
//     float *hostFloats2 = new float[N];
//     float *hostFloats3 = new float[N];
//     float *hostOut = new float[N];

//     struct Struct_fp_fp_f_f mystruct = {(float *)gpuFloats1, (float *)gpuFloats2, 3.0f, 8.0f};
//     struct_byvalue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, (float *)gpuOut);

//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cudaMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cudaMemcpy(hostOut, gpuOut, 4 * sizeof(float), cudaMemcpyDeviceToHost);

//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats2[0] << endl;
//     cout << hostOut[0] << endl;
//     cout << hostOut[1] << endl;

//     assert(hostFloats1[0] == 9);
//     assert(hostFloats2[0] == 10);
//     assert(hostOut[0] == 3);
//     assert(hostOut[1] == 8);

//     cudaFree(gpuFloats1);
//     cudaFree(gpuFloats2);
//     cudaFree(gpuFloats3);
//     cudaFree(gpuOut);

//     delete[]hostFloats1;
//     delete[]hostFloats2;
//     delete[]hostFloats3;
//     delete[]hostOut;

//     cuStreamDestroy(stream);
// }

// __global__ void struct_aspointer(struct Struct_2floats *mystruct, float *out) {
//     out[0] = mystruct->f1;
//     out[1] = mystruct->f2;
// }

// void testaspointerstruct() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostOut = new float[N];

//     float *gpuOut;
//     cudaMalloc((void**)(&gpuOut), N * sizeof(float));

//     struct Struct_2floats mystruct = { 5, 7 };
//     struct Struct_2floats *gpu_mystruct;
//     cudaMalloc((void**)(&gpu_mystruct), sizeof(mystruct));
//     cudaMemcpy(gpu_mystruct, &mystruct, sizeof(mystruct), cudaMemcpyHostToDevice);

//     struct_aspointer<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(gpu_mystruct, gpuOut);

//     cudaMemcpy(hostOut, gpuOut, 4 * sizeof(float), cudaMemcpyDeviceToHost);

//     cuStreamSynchronize(stream);

//     cout << hostOut[0] << endl;
//     cout << hostOut[1] << endl;

//     assert(hostOut[0] == 5);
//     assert(hostOut[1] == 7);

//     delete[]hostOut;

//     cuStreamDestroy(stream);
// }

__global__ void kernel_twostructs(struct Struct_fp_fp_f_f mystruct, struct Struct_fp mystruct2) {
    mystruct.p1[0] = 9.0f;
    mystruct.p2[0] = 10.0f;
    mystruct2.p1[0] = 11.0f;
}

void testtwostructs() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    float *gpuFloats2;
    hipMalloc((void**)(&gpuFloats2), N * sizeof(float));

    float *gpuFloats3;
    hipMalloc((void**)(&gpuFloats3), N * sizeof(float));

    float *hostFloats1 = new float[N];
    float *hostFloats2 = new float[N];
    float *hostFloats3 = new float[N];

    struct Struct_fp_fp_f_f mystruct = {(float *)gpuFloats1, (float *)gpuFloats2, 0.0f, 0.0f};
    struct Struct_fp mystruct2 = {(float *)gpuFloats3};

    kernel_twostructs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, mystruct2);

    hipMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats3, gpuFloats3, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;
    cout << hostFloats3[0] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);
    assert(hostFloats3[0] == 11);

    hipFree(gpuFloats1);
    hipFree(gpuFloats2);
    hipFree(gpuFloats3);

    delete[]hostFloats1;
    delete[]hostFloats2;
    delete[]hostFloats3;

    hipStreamDestroy(stream);
}

// __global__ void kernel_structbyval_noptrs(struct Struct_1float mystruct1, float *out) {
//     if(threadIdx.x == 0) {
//         out[0] = mystruct1.f1;
//         out[1] = 5;
//     }
// }

// void teststructbyvalNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1 = new float[N];

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     struct Struct_1float mystruct1 = {8.0f};

//     kernel_structbyval_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct1, (float *)gpuFloats1);

//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;

//     assert(hostFloats1[0] == 8);
//     assert(hostFloats1[1] == 5);

//     delete[] hostFloats1;
//     cudaFree(gpuFloats1);

//     cuStreamDestroy(stream);
// }

// __global__ void kernel_twostructs_noptrs(struct Struct_2floats *mystruct, struct Struct_1float *mystruct2, struct Struct_1float mystruct3, float *out) {
//     if(threadIdx.x == 0) {
//     out[0] = mystruct->f1;
//     out[1] = mystruct->f2;
//     out[2] = mystruct2->f1;
//     out[3] = mystruct3.f1;
//     }
// }

// void test_twostructs_byptr_NoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1 = new float[N];

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     struct Struct_2floats mystruct = {5.0f, 6.0f};
//     struct Struct_1float mystruct2 = {7.0f};
//     struct Struct_1float mystruct3 = {8.0f};

//     struct Struct_2floats *gpu_mystruct;
//     cudaMalloc((void**)(&gpu_mystruct), sizeof(mystruct));
//     cudaMemcpy(gpu_mystruct, &mystruct, sizeof(mystruct), cudaMemcpyHostToDevice);

//     struct Struct_1float *gpu_mystruct2;
//     cudaMalloc((void**)(&gpu_mystruct2), sizeof(mystruct2));
//     cudaMemcpy(gpu_mystruct2, &mystruct2, sizeof(mystruct2), cudaMemcpyHostToDevice);

//     kernel_twostructs_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(gpu_mystruct, gpu_mystruct2, mystruct3, (float *)gpuFloats1);
//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;
//     cout << hostFloats1[2] << endl;
//     cout << hostFloats1[3] << endl;

//     assert(hostFloats1[0] == 5);
//     assert(hostFloats1[1] == 6);
//     assert(hostFloats1[2] == 7);
//     assert(hostFloats1[3] == 8);

//     cudaFree(gpuFloats1);
//     cudaFree(gpu_mystruct);
//     cudaFree(gpu_mystruct2);
//     // cudaFree(gpu_mystruct3);

//     delete[] hostFloats1;

//     cuStreamDestroy(stream);
// }

// __global__ void kernel_struct2byval_noptrs(struct Struct_2floats mystruct1, float *out) {
//     if(threadIdx.x == 0) {
//     out[0] = mystruct1.f1;
//     out[1] = mystruct1.f2;
//     }
// }

// void teststruct2byvalNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1 = new float[N];

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     struct Struct_2floats mystruct1 = {8.0f, 9.0f};

//     kernel_struct2byval_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct1, (float *)gpuFloats1);
//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;

//     assert(hostFloats1[0] == 8);
//     assert(hostFloats1[1] == 9);

//     delete[] hostFloats1;
//     cudaFree(gpuFloats1);

//     cuStreamDestroy(stream);
// }

// struct struct_f_c_f_c {
//     float f1;
//     char c1;
//     float f2;
//     char c2;
// };

// __global__ void kernel_twostructs_gpuside_singlebuffer(struct struct_f_c_f_c *mystruct1, struct struct_f_c_f_c *mystruct2, float *out) {
//     out[0] = mystruct1->f1;
//     out[1] = mystruct1->f2;
//     out[2] = mystruct2->f1;
//     out[3] = mystruct2->f2;
// }

// void test_twostructs_gpuside_singlebuffer() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostOut = new float[N];

//     float *gpuOut;
//     cudaMalloc((void**)(&gpuOut), N * sizeof(float));

//     char *gpubuf;
//     cudaMalloc((void **)&gpubuf, 1024);

//     int offset1 = 24;
//     int offset2 = 40;

//     struct struct_f_c_f_c mystruct1 = { 5, 0, 7, 0 };
//     cudaMemcpy(gpubuf + offset1, &mystruct1, sizeof(mystruct1), cudaMemcpyHostToDevice);

//     struct struct_f_c_f_c mystruct2 = { 9, 0, 3, 0 };
//     cudaMemcpy(gpubuf + offset2, &mystruct2, sizeof(mystruct2), cudaMemcpyHostToDevice);

//     kernel_twostructs_gpuside_singlebuffer<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(
//         (struct struct_f_c_f_c *)(gpubuf + offset1),
//         (struct struct_f_c_f_c *)(gpubuf + offset2),
//         gpuOut);

//     cudaMemcpy(hostOut, gpuOut, 4 * sizeof(float), cudaMemcpyDeviceToHost);

//     cuStreamSynchronize(stream);

//     cout << hostOut[0] << endl;
//     cout << hostOut[1] << endl;
//     cout << hostOut[2] << endl;
//     cout << hostOut[3] << endl;

//     assert(hostOut[0] == 5);
//     assert(hostOut[1] == 7);
//     assert(hostOut[2] == 9);
//     assert(hostOut[3] == 3);

//     delete[]hostOut;

//     cuStreamDestroy(stream);
// }

// struct NestL2 {
//     float floats[10];
// };

// struct NestL1 {
//     struct NestL2 n1;
//     struct NestL2 n2;
// };

// struct NestTop {
//     struct NestL1 n1;
//     struct NestL1 n2;
// };

// __global__ void kernelUseNestTop(NestTop nest, float *out) {
//     out[0] = nest.n1.n1.floats[0];
//     out[1] = nest.n1.n1.floats[1];
// }

// void testKernelUsesNestTop() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *gpuOut;
//     cudaMalloc((void**)(&gpuOut), N * sizeof(float));

//     float *hostOut = new float[N];

//     struct NestTop nestTop;
//     nestTop.n1.n1.floats[0] = 5;
//     nestTop.n1.n1.floats[1] = 7;

//     kernelUseNestTop<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(nestTop, (float *)gpuOut);

//     cudaMemcpy(hostOut, gpuOut, 4 * sizeof(float), cudaMemcpyDeviceToHost);

//     cuStreamSynchronize(stream);

//     cout << hostOut[0] << endl;
//     cout << hostOut[1] << endl;

//     assert(hostOut[0] == 5);
//     assert(hostOut[1] == 7);

//     cudaFree(gpuOut);

//     delete[]hostOut;

//     cuStreamDestroy(stream);    
// }

struct NestL2 {
    float floats[10];
};

struct NestL1 {
    struct NestL2 n1;
    struct NestL2 n2;
};

struct NestTop {
    struct NestL1 n1;
    struct NestL1 n2;
};

__global__ void kernelUseNestTop(NestTop nest, float *out) {
    out[0] = nest.n1.n1.floats[0];
    out[1] = nest.n1.n1.floats[1];
}

void testKernelUsesNestTop() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *gpuOut;
    hipMalloc((void**)(&gpuOut), N * sizeof(float));

    float *hostOut = new float[N];

    struct NestTop nestTop;
    nestTop.n1.n1.floats[0] = 5;
    nestTop.n1.n1.floats[1] = 7;

    kernelUseNestTop<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(nestTop, (float *)gpuOut);

    hipMemcpy(hostOut, gpuOut, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostOut[0] << endl;
    cout << hostOut[1] << endl;

    assert(hostOut[0] == 5);
    assert(hostOut[1] == 7);

    hipFree(gpuOut);

    delete[]hostOut;

    hipStreamDestroy(stream);    
}

__global__ void struct_byvalue_withreadnone(struct Struct_fp_fp_f_f mystruct, struct Struct_fp_fp_f_f donothing, float *out) {
    out[0] = mystruct.f1;
    out[1] = mystruct.f2;
    mystruct.p1[0] = 9.0f;
    mystruct.p2[0] = 10.0f;
}

void testbyvaluestruct_withreadnone() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    float *gpuFloats2;
    hipMalloc((void**)(&gpuFloats2), N * sizeof(float));

    float *gpuFloats3;
    hipMalloc((void**)(&gpuFloats3), N * sizeof(float));

    float *gpuOut;
    hipMalloc((void**)(&gpuOut), N * sizeof(float));

    float *hostFloats1 = new float[N];
    float *hostFloats2 = new float[N];
    float *hostFloats3 = new float[N];
    float *hostOut = new float[N];

    struct Struct_fp_fp_f_f mystruct = {(float *)gpuFloats1, (float *)gpuFloats2, 3.0f, 8.0f};
    struct Struct_fp_fp_f_f donothing = {(float *)0, (float *)0, 0.0f, 0.0f};
    struct_byvalue_withreadnone<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, donothing, (float *)gpuOut);

    hipMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostOut, gpuOut, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;
    cout << hostOut[0] << endl;
    cout << hostOut[1] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);
    assert(hostOut[0] == 3);
    assert(hostOut[1] == 8);

    hipFree(gpuFloats1);
    hipFree(gpuFloats2);
    hipFree(gpuFloats3);
    hipFree(gpuOut);

    delete[]hostFloats1;
    delete[]hostFloats2;
    delete[]hostFloats3;
    delete[]hostOut;

    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    // cout << "\ntestvaluestruct" << endl;
    // testbyvaluestruct();

    // cout << "\ntestaspointersstruct" << endl;
    // testaspointerstruct();

    cout << "\ntesttwostructs" << endl;
    testtwostructs();

    // cout << "\nteststructbyvalNoPtr" << endl;
    // teststructbyvalNoPtr();

    // cout << "\ntest_twostructs_byptr_NoPtr" << endl;
    // test_twostructs_byptr_NoPtr();

    // cout << "\nteststruct2byvalNoPtr" << endl;
    // teststruct2byvalNoPtr();

    // cout << "\ntest_twostructs_gpuside_singlebuffer" << endl;
    // test_twostructs_gpuside_singlebuffer();

    // cout << "\ntestKernelUsesNestTop" << endl;
    // testKernelUsesNestTop();

    cout << "\ntestKernelUsesNestTop" << endl;
    testKernelUsesNestTop();

    cout << "\ntestvaluestruct_withreadnone" << endl;
    testbyvaluestruct_withreadnone();

    return 0;
}
