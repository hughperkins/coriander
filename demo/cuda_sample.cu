// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;
    float *gpuFloats;
    float hostFloats[4];

    // allocate GPU memory
    hipMalloc((void**)(&gpuFloats), N * sizeof(float));

    // launch kernel
    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats, 2, 123.0f);

    // copy from gpu to host
    hipMemcpy(hostFloats, gpuFloats, 4 * sizeof(float), hipMemcpyDeviceToHost);

    cout << "hostFloats[2] " << hostFloats[2] << endl;

    hipFree(gpuFloats);

    return 0;
}
