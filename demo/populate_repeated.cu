#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void populateArray(int *array, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        array[idx] = idx * 7;
    }
}

int main() {
    const int N = 1e9;
    int *h_array = new int[N];
    int *d_array;

    hipMalloc((void **)&d_array, N * sizeof(int));

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    auto start = std::chrono::high_resolution_clock::now();
    while(true) {
        populateArray<<<gridSize, blockSize>>>(d_array, N);
        std::cout << "." << std::flush;
    }
    hipMemcpy(h_array, d_array, N * sizeof(int), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Elapsed time: " << duration.count() / 1000 << " milliseconds\n";

    std::cout << "First 10 elements of the array:\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << "array[" << i << "] = " << h_array[i] << "\n";
    }

    delete[] h_array;
    hipFree(d_array);

    return 0;
}
