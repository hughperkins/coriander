#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void pseudorandomTransformKernel(int* data, int N, int steps) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < N) {
        unsigned int state = idx;
        
        // Linear Congruential Generator (LCG)
        for (int i = 0; i < steps; i++) {
            state = (1664525 * state + 1013904223) % 4294967295;
        }
        
        data[idx] = (int)state % 10000;
    }
}

int main() {
    const int N = 1e8;
    const int steps = 5000;
    int *h_array = new int[N];
    int *d_array;

    hipMalloc((void **)&d_array, N * sizeof(int));

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    auto start = std::chrono::high_resolution_clock::now();
    pseudorandomTransformKernel<<<gridSize, blockSize>>>(d_array, N, steps);
    hipMemcpy(h_array, d_array, N * sizeof(int), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Elapsed time: " << duration.count() / 1000 << " milliseconds\n";

    std::cout << "First 10 elements of the array:\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << "array[" << i << "] = " << h_array[i] << "\n";
    }

    delete[] h_array;
    hipFree(d_array);

    return 0;
}
